#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float4, 2>   result_buffer;

rtDeclareVariable(float3,                draw_color, , );

RT_PROGRAM void draw_solid_color()
{
  result_buffer[launch_index] = make_float4(draw_color, 0.f);
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////   Adaptive Additional Rays Test   //////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

rtDeclareVariable(float3,		// type of the variable used in an RT_PROGRAM in this .cu file
	shading_normal,				// name of the variable used in an RT_PROGRAM in this .cu file
	attribute shading_normal,	// semantic variable declared on the API in the corresponding .cpp file, attribute is to specify that the variable is part of a struct
	);

// per ray data struct
struct PerRayData_radiance
{
	float3 result;				// struct variable carrying our calculated output
	float  importance;
	int depth;
	int done;
};

rtDeclareVariable(PerRayData_radiance,
	prd_radiance, 
	rtPayload,							//This is a semantic nam, not an API declared variable name to bind user data to
	);

RT_PROGRAM void closest_hit_radiance0()
{
	prd_radiance.result = normalize(rtTransformNormal(	// transforms n as a normal using the current active transformation stack (the inverse transpose)
		RT_OBJECT_TO_WORLD,								// other option would be RT_WORLD_TO_OBJECT
		shading_normal))
		*0.5f + 0.5f;
	prd_radiance.done = true;
}

rtDeclareVariable(float3, bg_color, , );

// Miss program for a ray, in case a ray misses the geometry give it the background color.
RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;
	prd_radiance.done = true;
}

static __device__ __inline__ uchar4 make_color(const float3& c)
{
	return make_uchar4(static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* B */
	static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
	static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* R */
	255u);                                                 /* A */
	};


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtBuffer<uchar4, 2>   output_buffer;

/* Aaptive additional rays variables */
rtDeclareVariable(uint, max_per_launch_idx_ray_budget, , ) = static_cast<uint>(5u);		/* this variable will be written by the user */
rtBuffer<uchar4, 2>   additional_rays_buffer;				/* this buffer will be initialized by the host, but must also be modified by the graphics device */

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) /
		make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);
	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;
	prd.done = false;

	rtTrace(top_object, ray, prd); /* find out when its done, its important to know whether the code proceeds after this line after "rtTrace" is 'finished', 
								   or if it starts a parallel subroutine and the code advances without waiting for "rtTrace" to finish (i assume the latter, 
								   due to what i read in the technical overview -> the former is true, evidence by testing) */

	output_buffer[launch_index] = make_color(prd.result);
	
	/* Testing for additional adaptive rays */
	
	/* 
		Postpone launching additional rays until first currently traced ray output is avaible (extend to neighborhood after success).
			- 1. Postponing will be done with a loop, which will run indefinitely and does nothing (maybe use observer pattern here, more elegant than having a loop with an if statement), 
				 until a condition is met, in this case when the output buffer has been written (-> no longer necessary, because the code advances after "rtTrace" only after its done).
			  2. Upon reaching the written output buffer state which i will modify the additional "additional_rays_buffer" values, which are initialized with ("max_per_launch_idx_ray_budget" + 1)
			     so that they contain an arbitary smaller or value (but only corresponding (neighboring) values to the current launchIdx).
			  3. After setting the current additional(, adaptive) ray budget i break/leave the loop and start another, that launches another loop, in which i launch additional rays,
			     according to the current budget and add/write the results into the output buffer.
		Additional adaptive rays count will be avaible in the "additional_rays_buffer"
	*/
	
	uint additional_rays_count = static_cast<uint>(additional_rays_buffer[launch_index].x);

	/* Make the following 'adaptive pass' test to a real adaptive pass (for that i must ensure, that the first resulting image is completely avaible). */
	if (prd.done)
	{
		additional_rays_count = static_cast<uint>(output_buffer[launch_index].x) % (max_per_launch_idx_ray_budget + 1u);
		//rtPrintf("Launch index: %u, %u; Additional rays count: %u !\n\n", launch_index.x, launch_index.y, additional_rays_count);
		float jitter = static_cast<float>(additional_rays_count) / static_cast<float>(max_per_launch_idx_ray_budget);
		float jitterScale = 0.1f;
		jitter = jitter * jitterScale;
		while (additional_rays_count > 0u)
		{
			//rtPrintf("Additional rays left: %u !\n", additional_rays_count);
			float3 jittered_ray_origin;

			jittered_ray_origin.x = ray_origin.x + jitter;
			jittered_ray_origin.y = ray_origin.y - jitter;
			jittered_ray_origin.z = ray_origin.z + jitter;

			float3 jittered_ray_direction;

			jittered_ray_direction.x = ray_direction.x + jitter;
			jittered_ray_direction.y = ray_direction.y - jitter;
			jittered_ray_direction.z = ray_direction.z + jitter;

			Ray ray2(jittered_ray_origin, jittered_ray_direction, radiance_ray_type, scene_epsilon);
			PerRayData_radiance prd2;
			prd2.importance = 1.f;
			prd2.depth = 0;
			prd2.done = false;

			rtTrace(top_object, ray2, prd2);

			output_buffer[launch_index] = make_color(prd2.result);
			additional_rays_count--;

			jitter = static_cast<float>(additional_rays_count) / static_cast<float>(max_per_launch_idx_ray_budget);
			jitterScale = jitterScale * -1.f;
			jitter = jitter * jitterScale;
		}
	}
}

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}