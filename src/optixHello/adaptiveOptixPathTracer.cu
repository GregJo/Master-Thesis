#include "hip/hip_runtime.h"
/*
* Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"
#include "random.h"

//#include "VarianceAdaptive.h"
#include "HoelderAdaptive.h"
#include "MitchellFilterDevice.h"

using namespace optix;

struct PerRayData_pathtrace
{
	float3 result;
	float3 radiance;
	float3 attenuation;
	float3 origin;
	float3 direction;
	unsigned int seed;
	int depth;
	int countEmitted;
	int done;
	//int isAdaptive;
};

struct PerRayData_pathtrace_shadow
{
	bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, far_plane, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(unsigned int, frame_number, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );
rtDeclareVariable(unsigned int, pathtrace_ray_type, , );

RT_PROGRAM void pathtrace_camera_adaptive()
{
	//rtPrintf("Current samples number: %d\n\n", adaptive_samples_budget_buffer[launch_index].x);

	// Debug!
	depth_gradient_buffer[launch_index] = make_float4(0.0f);

	/*size_t2 screen = hoelder_adaptive_buffers[input_scene_render_buffer].size();*/
	size_t2 screen = output_buffer.size();

	float2 inv_screen = 1.0f / make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

	float2 jitter_scale = inv_screen / sqrt_num_samples;

	initializeHoelderAdaptiveSceneDepthBuffer(launch_index, frame_number, camera_changed);
	initialize_hoelder_refinement_buffer(launch_index, frame_number, camera_changed, static_cast<uint>(window_size));

	unsigned int adaptive_samples_per_pixel = compute_current_samples_number(launch_index, window_size_buffer[launch_index].x);
	unsigned int current_samples_per_pixel = adaptive_samples_per_pixel;
	float3 result = make_float3(0.0f);

	unsigned int adaptive_sqrt_num_samples = sqrtf(static_cast<float>(adaptive_samples_per_pixel));

	if (!adaptive_sqrt_num_samples)
	{
		++adaptive_sqrt_num_samples;
	}

	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);

	//float3 pixel_color = make_float3(hoelder_adaptive_buffers[input_scene_render_buffer][launch_index]);

	//resetHoelderAdaptiveSceneDepthBuffer(launch_index);

	if (current_samples_per_pixel)
	{
		//post_process_output_buffer[launch_index] = hoelder_adaptive_buffers[input_scene_render_buffer][launch_index];
		do
		{
			//
			// Sample pixel using jittering
			//
			unsigned int x = adaptive_samples_per_pixel % adaptive_sqrt_num_samples;
			unsigned int y = adaptive_samples_per_pixel / adaptive_sqrt_num_samples;
			float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
			float2 d = pixel + jitter*jitter_scale;
			float3 ray_origin = eye;
			float3 ray_direction = normalize(d.x*U + d.y*V + W);

			//rtPrintf("2D ray direction in screen coordinates: [ %f , %f ]\n", d.x, d.y);

			// Initialze per-ray data
			PerRayData_pathtrace prd;
			prd.result = make_float3(0.f);
			prd.attenuation = make_float3(1.f);
			prd.countEmitted = true;
			prd.done = false;
			prd.seed = seed;
			prd.depth = 0;
			//prd.isAdaptive = 1;

			// Each iteration is a segment of the ray path.  The closest hit will
			// return new segments to be traced here.
			for (;;)
			{
				if (prd.depth == 1)
				{
					float ray_length = fabsf(length((prd.origin - eye)));
					float normalized_ray_length = ray_length / far_plane;//2500.0f;

					float a = 1.0f / (float)frame_number;
					float3 old_depth = make_float3(output_scene_depth_buffer[launch_index]);
					output_scene_depth_buffer[launch_index] = make_float4(lerp(old_depth, make_float3(normalized_ray_length), a), 1.0f);

					hoelder_adaptive_scene_depth_buffer[launch_index] = make_float4(make_float3(normalized_ray_length), 1.0f);
					//if (frame_number == 1)
					//{
					//	input_scene_depth_buffer[launch_index] = make_float4(normalized_ray_length);
					//}
				}

				//if (prd.depth == 1)
				//{
				//	float ray_length = fabsf(length((prd.origin - eye)));
				//	float normalized_ray_length = ray_length / far_plane;//2500.0f;

				//	hoelder_adaptive_scene_depth_buffer[launch_index] = make_float4(make_float3(normalized_ray_length), 1.0f);
				//}

				Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(top_object, ray, prd);

				if (prd.done)
				{
					// We have hit the background or a luminaire
					prd.result += prd.radiance * prd.attenuation;
					break;
				}

				// Russian roulette termination 
				if (prd.depth >= rr_begin_depth)
				{
					float pcont = fmaxf(prd.attenuation);
					if (rnd(prd.seed) >= pcont)
						break;
					prd.attenuation /= pcont;
				}

				prd.depth++;
				prd.result += prd.radiance * prd.attenuation;

				// Update ray data for the next path segment
				ray_origin = prd.origin;
				ray_direction = prd.direction;
			}

			//result += prd.result;
			float2 sample = make_float2(d.x + launch_index.x, d.y + launch_index.y);

			//float reconstruction_filter_weight = computeMitchellFilterSampleContribution(sample, launch_index);

			//computeMitchellFilterSampleContributionInNeighborhood(sample, launch_index, prd.result, current_samples_per_pixel, screen, &input_current_total_rays_buffer, &post_process_output_buffer);

			//result += prd.result * reconstruction_filter_weight;

			//result += prd.result * reconstruction_filter_weight * 1.0f / input_current_total_rays_buffer[launch_index].x;

			seed = prd.seed;
			output_current_total_rays_buffer[launch_index].x++;
			//rtPrintf("Launch index: [ %d , %d ], current total samples: [ %d ]\n\n", launch_index.x, launch_index.y, input_current_total_rays_buffer[launch_index].x);

			//if (launch_index.x == 256 && launch_index.y == 256)// || normalized_dist_length > 0.5f)
			//{
			//	rtPrintf("\nTotal rays: %d\n", input_current_total_rays_buffer[launch_index].x);
			//}
			int current_total_rays = output_current_total_rays_buffer[launch_index].x;
			computeMitchellFilterSampleContributionInNeighborhood(sample, launch_index, prd.result, screen, current_total_rays, &output_filter_sum_buffer, &output_filter_x_sample_sum_buffer);
		} while (--current_samples_per_pixel);

		//float3 old_color = make_float3(hoelder_adaptive_buffers[input_scene_render_buffer][launch_index]);
		//pixel_color = (1.0f - static_cast<float>(adaptive_samples_per_pixel) / input_current_total_rays_buffer[launch_index].x) * old_color 
		//	+ (static_cast<float>(adaptive_samples_per_pixel) / input_current_total_rays_buffer[launch_index].x) * result;// / (adaptive_sqrt_num_samples*adaptive_sqrt_num_samples);

		// Pink coloring of tiles for debug
		//if (adaptive_samples_per_pixel == 1 && window_size_buffer[launch_index].x <= 4)
		//{
		//	pixel_color = make_float3(window_size, 0.0f, window_size_buffer[launch_index].x);
		//}

		//if (adaptive_samples_per_pixel >= 1)
		//{
		//	pixel_color = make_float3(0.0f);
		//}
	}
	//
	// Update the output buffer
	//

	//float a = 1.0f / (float)frame_number;
	//float3 old_color = make_float3(hoelder_adaptive_buffers[input_scene_render_buffer][launch_index]);
	//post_process_output_buffer[launch_index] = make_float4(pixel_color, 1.0f);//make_float4(lerp(old_color, pixel_color, a), 1.0f);
	//evaluatePixelFileringEquation(launch_index, &post_process_output_buffer, &input_filter_sum_buffer, &input_filter_x_sample_sum_buffer);
	evaluatePixelFileringEquation(launch_index, &output_buffer, &output_filter_sum_buffer, &output_filter_x_sample_sum_buffer);

	//compute_current_window_test(launch_index, 5);
}

//
// Adaptive version of pathtracing end
//
