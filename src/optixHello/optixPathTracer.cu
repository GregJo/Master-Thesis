#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"
#include "random.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
	//int isAdaptive;
};

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(float,		 far_plane, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );

rtBuffer<float4, 2>              output_buffer;
rtBuffer<ParallelogramLight>     lights;

// Adaptive post processing variables and buffers
rtBuffer<int4, 2>				 additional_rays_buffer_input;										/* this buffer will be initialized by the host, but must also be modified by the graphics device */
rtBuffer<float4, 2>              per_window_variance_buffer_input;
rtBuffer<float4, 2>              output_scene_depth_buffer;
rtBuffer<float4, 2>				 post_process_input_buffer;
rtBuffer<float4, 2>				 post_process_input_scene_depth_buffer;

rtDeclareVariable(unsigned int, window_size, , );
rtDeclareVariable(unsigned int, max_ray_budget_total, , ) = static_cast<uint>(50u);
rtDeclareVariable(unsigned int, max_per_launch_idx_ray_budget, , ) = static_cast<uint>(5u);		/* this variable will be written by the user */
rtDeclareVariable(int, camera_changed, , );

static __device__ __inline__ void reset_additional_rays_buffer(uint2 current_launch_index)
{
	additional_rays_buffer_input[current_launch_index] = make_int4(static_cast<int>(max_ray_budget_total));
};

static __device__ __inline__ uint2 compute_variance_window_center(uint2 current_launch_index, uint window_size)
{
	size_t2 screen = output_buffer.size();

	uint times_width = screen.x / window_size;
	uint times_height = screen.y / window_size;

	uint horizontal_padding = static_cast<uint>((screen.x - (times_width * window_size)) / 2);
	uint vertical_padding = static_cast<uint>((screen.y - (times_height * window_size)) / 2);

	uint half_window_size = (window_size / 2) + (window_size % 2);

	uint2 times_launch_index = make_uint2(((current_launch_index.x / window_size) * window_size) % screen.x, ((current_launch_index.y / window_size) * window_size) % screen.y);

	uint2 current_window_center = make_uint2(times_launch_index.x + horizontal_padding + half_window_size, times_launch_index.y + vertical_padding + half_window_size);

	return current_window_center;
};

RT_PROGRAM void pathtrace_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);

	if (camera_changed == 1)
	{
		//rtPrintf("Reset additional rays buffer!!!\n\n");
		output_scene_depth_buffer[launch_index] = make_float4(1.0f, 1.0f, 1.0f, 1.0f);
		reset_additional_rays_buffer(launch_index);
	}

	if (camera_changed == 0 && frame_number > 1)
	{
		//rtPrintf("Using postprocess values!!!\n\n");
		output_buffer[launch_index] = post_process_input_buffer[launch_index];
		output_scene_depth_buffer[launch_index] = post_process_input_scene_depth_buffer[launch_index];
	}
	//else
	//{
		do
		{
			//
			// Sample pixel using jittering
			//
			unsigned int x = samples_per_pixel%sqrt_num_samples;
			unsigned int y = samples_per_pixel / sqrt_num_samples;
			float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
			float2 d = pixel + jitter*jitter_scale;
			float3 ray_origin = eye;
			float3 ray_direction = normalize(d.x*U + d.y*V + W);

			// Initialze per-ray data
			PerRayData_pathtrace prd;
			prd.result = make_float3(0.f);
			prd.attenuation = make_float3(1.f);
			prd.countEmitted = true;
			prd.done = false;
			prd.seed = seed;
			prd.depth = 0;
			//prd.isAdaptive = 0;

			// Each iteration is a segment of the ray path.  The closest hit will
			// return new segments to be traced here.
			for (;;)
			{
				if (prd.depth == 1)
				{
					float ray_length = fabsf(length((prd.origin - eye)));
					float normalized_ray_length = ray_length / far_plane;//2500.0f;

					float a = 1.0f / (float)frame_number;
					float3 old_depth = make_float3(output_scene_depth_buffer[launch_index]);
					output_scene_depth_buffer[launch_index] = make_float4(lerp(old_depth, make_float3(normalized_ray_length), a), 1.0f);

					//output_scene_depth_buffer[launch_index] = 0.9f * output_scene_depth_buffer[launch_index] + 0.1f * make_float4(normalized_ray_length);
					if (frame_number == 1)
					{
						output_scene_depth_buffer[launch_index] = make_float4(normalized_ray_length);
					}
				}

				Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(top_object, ray, prd);

				if (prd.done)
				{
					// We have hit the background or a luminaire
					prd.result += prd.radiance * prd.attenuation;
					break;
				}

				// Russian roulette termination 
				if (prd.depth >= rr_begin_depth)
				{
					float pcont = fmaxf(prd.attenuation);
					if (rnd(prd.seed) >= pcont)
						break;
					prd.attenuation /= pcont;
				}

				prd.depth++;
				prd.result += prd.radiance * prd.attenuation;

				// Update ray data for the next path segment
				ray_origin = prd.origin;
				ray_direction = prd.direction;
			}

			result += prd.result;
			seed = prd.seed;
		} while (--samples_per_pixel);

		//
		// Update the output buffer
		//
		float3 pixel_color = result / (sqrt_num_samples*sqrt_num_samples);

		if (frame_number > 1)
		{
			float a = 1.0f / (float)frame_number;
			float3 old_color = make_float3(output_buffer[launch_index]);
			output_buffer[launch_index] = make_float4(lerp(old_color, pixel_color, a), 1.0f);
		}
		else
		{
			output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
		}
	//}
	per_window_variance_buffer_input[compute_variance_window_center(launch_index, window_size)] = make_float4(-1.0f);
}


//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}


//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,     diffuse_color, , );

//
// Diffuse texture and sampler
//
rtTextureSampler<float4, 2> Kd_map;
//rtTextureSampler<float4, 2> Ks_map;		// specular
rtTextureSampler<float4, 2> D_map;		// alpha texture
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );

RT_PROGRAM void diffuseTextured()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hitpoint = ray.origin + t_hit * ray.direction;

	//if (current_prd.depth == 0 && current_prd.isAdaptive == 0)
	//{
	//	float ray_length = fabsf(length((hitpoint - eye)));
	//	float normalized_ray_length = ray_length / 2500.0f;
	//	output_scene_depth_buffer[launch_index] = make_float4(normalized_ray_length);
	//}

	//
	// Generate a reflection ray.  This will be traced back in ray-gen.
	//
	current_prd.origin = hitpoint;

	float z1 = rnd(current_prd.seed);
	float z2 = rnd(current_prd.seed);
	float3 p;
	cosine_sample_hemisphere(z1, z2, p);
	optix::Onb onb(ffnormal);
	onb.inverse_transform(p);
	current_prd.direction = p;

	// Diffuse texture value
	const float3 diffuse_tex_sample = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y));
	const float3 alpha_tex_sample = make_float3(tex2D(D_map, texcoord.x, texcoord.y));

	//if (alpha_tex_sample.x != 0.0f)
	//{
		// NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
		// with cosine density.
		current_prd.attenuation = current_prd.attenuation * diffuse_tex_sample;
		current_prd.countEmitted = false;

		//
		// Next event estimation (compute direct lighting).
		//
		unsigned int num_lights = lights.size();
		float3 result = make_float3(0.0f);

		for (int i = 0; i < num_lights; ++i)
		{
			// Choose random point on light
			ParallelogramLight light = lights[i];
			const float z1 = rnd(current_prd.seed);
			const float z2 = rnd(current_prd.seed);
			const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

			// Calculate properties of light sample (for area based pdf)
			const float  Ldist = length(light_pos - hitpoint);
			const float3 L = normalize(light_pos - hitpoint);
			const float  nDl = dot(ffnormal, L);
			const float  LnDl = dot(light.normal, L);

			// cast shadow ray
			if (nDl > 0.0f && LnDl > 0.0f)
			{
				PerRayData_pathtrace_shadow shadow_prd;
				shadow_prd.inShadow = false;
				// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
				Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
				rtTrace(top_object, shadow_ray, shadow_prd);

				if (!shadow_prd.inShadow)
				{
					const float A = length(cross(light.v1, light.v2));
					// convert area based pdf to solid angle
					const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
					result += light.emission * weight;
				}
			}
		}

		current_prd.radiance = result;
	//}
}

//RT_PROGRAM void diffuse()
//{
//    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
//    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
//    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
//
//    float3 hitpoint = ray.origin + t_hit * ray.direction;
//
//    //
//    // Generate a reflection ray.  This will be traced back in ray-gen.
//    //
//    current_prd.origin = hitpoint;
//
//    float z1=rnd(current_prd.seed);
//    float z2=rnd(current_prd.seed);
//    float3 p;
//    cosine_sample_hemisphere(z1, z2, p);
//    optix::Onb onb( ffnormal );
//    onb.inverse_transform( p );
//    current_prd.direction = p;
//
//    // NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
//    // with cosine density.
//    current_prd.attenuation = current_prd.attenuation * diffuse_color;
//    current_prd.countEmitted = false;
//
//    //
//    // Next event estimation (compute direct lighting).
//    //
//    unsigned int num_lights = lights.size();
//    float3 result = make_float3(0.0f);
//
//    for(int i = 0; i < num_lights; ++i)
//    {
//        // Choose random point on light
//        ParallelogramLight light = lights[i];
//        const float z1 = rnd(current_prd.seed);
//        const float z2 = rnd(current_prd.seed);
//        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;
//
//        // Calculate properties of light sample (for area based pdf)
//        const float  Ldist = length(light_pos - hitpoint);
//        const float3 L     = normalize(light_pos - hitpoint);
//        const float  nDl   = dot( ffnormal, L );
//        const float  LnDl  = dot( light.normal, L );
//
//        // cast shadow ray
//        if ( nDl > 0.0f && LnDl > 0.0f )
//        {
//            PerRayData_pathtrace_shadow shadow_prd;
//            shadow_prd.inShadow = false;
//            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
//            Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
//            rtTrace(top_object, shadow_ray, shadow_prd);
//
//            if(!shadow_prd.inShadow)
//            {
//                const float A = length(cross(light.v1, light.v2));
//                // convert area based pdf to solid angle
//                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
//                result += light.emission * weight;
//            }
//        }
//    }
//
//    current_prd.radiance = result;
//}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
	const float3 alpha_tex_sample = make_float3(tex2D(D_map, texcoord.x, texcoord.y));
	if (alpha_tex_sample.x == 0.0f)
	{
		rtIgnoreIntersection();
	}
	else
	{
		current_prd_shadow.inShadow = true;
		rtTerminateRay();
	}
}

RT_PROGRAM void any_hit_radiance()
{
	const float3 alpha_tex_sample = make_float3(tex2D(D_map, texcoord.x, texcoord.y));
	if (alpha_tex_sample.x == 0.0f)
	{
		rtIgnoreIntersection();
	}
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void miss()
{
    current_prd.radiance = bg_color;
    current_prd.done = true;
}


