#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"
#include "random.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );

rtBuffer<float4, 2>              output_buffer;
rtBuffer<ParallelogramLight>     lights;


RT_PROGRAM void pathtrace_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
    do 
    {
        //
        // Sample pixel using jittering
        //
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        float2 d = pixel + jitter*jitter_scale;
        float3 ray_origin = eye;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.f);
        prd.countEmitted = true;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(;;)
        {
            Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);

            if(prd.done)
            {
                // We have hit the background or a luminaire
                prd.result += prd.radiance * prd.attenuation;
                break;
            }

            // Russian roulette termination 
            if(prd.depth >= rr_begin_depth)
            {
                float pcont = fmaxf(prd.attenuation);
                if(rnd(prd.seed) >= pcont)
                    break;
                prd.attenuation /= pcont;
            }

            prd.depth++;
            prd.result += prd.radiance * prd.attenuation;

            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += prd.result;
        seed = prd.seed;
    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }
}


//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}


//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,     diffuse_color, , );

//
// Diffuse texture and sampler
//
rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );

RT_PROGRAM void diffuseTextured()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hitpoint = ray.origin + t_hit * ray.direction;

	//
	// Generate a reflection ray.  This will be traced back in ray-gen.
	//
	current_prd.origin = hitpoint;

	float z1 = rnd(current_prd.seed);
	float z2 = rnd(current_prd.seed);
	float3 p;
	cosine_sample_hemisphere(z1, z2, p);
	optix::Onb onb(ffnormal);
	onb.inverse_transform(p);
	current_prd.direction = p;

	// Diffuse texture value
	const float3 diffuse_tex_sample = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y));

	// NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
	// with cosine density.
	current_prd.attenuation = current_prd.attenuation * diffuse_tex_sample;
	current_prd.countEmitted = false;

	//
	// Next event estimation (compute direct lighting).
	//
	unsigned int num_lights = lights.size();
	float3 result = make_float3(0.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float z1 = rnd(current_prd.seed);
		const float z2 = rnd(current_prd.seed);
		const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot(light.normal, L);

		// cast shadow ray
		if (nDl > 0.0f && LnDl > 0.0f)
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
			Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_object, shadow_ray, shadow_prd);

			if (!shadow_prd.inShadow)
			{
				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
				result += light.emission * weight;
			}
		}
	}

	current_prd.radiance = result;
}

//
// Adaptive version of pathtracing begin
//

/*--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------*/
/* Adaptive additional rays variables */
rtDeclareVariable(unsigned int, max_per_launch_idx_ray_budget, , ) = static_cast<uint>(5u);		/* this variable will be written by the user */
//rtDeclareVariable(int, additional_sample_map_written, , ) = 0;
//rtBuffer<uchar4, 2>   additional_rays_buffer;										/* this buffer will be initialized by the host, but must also be modified by the graphics device */

rtBuffer<float4, 2>   input_buffer;														/* this buffer contains the initially rendered picture to be post processed */
rtBuffer<float4, 2>   post_process_output_buffer;										/* this buffer contains the result, processed with additional adaptive rays */

rtDeclareVariable(float, window_size, , );

static __device__ __inline__ float compute_window_variance(uint2 center, uint window_size)
{
	size_t2 screen = input_buffer.size();

	float mean = 0.f;
	float variance = 0.f;
	uint squared_window_size = window_size * window_size;
	uint half_window_size = (window_size / 2) + (window_size % 2);
	uint2 top_left_window_corner = make_uint2(center.x - half_window_size, center.y - half_window_size);

	//rtPrintf("\nTop left window corner: [ %d, %d ]\n", top_left_window_corner.x, top_left_window_corner.y);

	/* compute mean value */
	for (uint i = 0; i < squared_window_size; i++)
	{
		uint2 idx = make_uint2((i % window_size + top_left_window_corner.x) % screen.x, (i / window_size + top_left_window_corner.y) % screen.y);
		float3 input_buffer_val = make_float3(input_buffer[idx].x, input_buffer[idx].y, input_buffer[idx].z);
		mean += 1.f/3.f * (input_buffer_val.x + input_buffer_val.y + input_buffer_val.z);
		//if (center.x + center.y < 20)
		//{
		//	//rtPrintf("Current 1D index: %d\n", i);
		//	//rtPrintf("Current relative 2D index: [ %d, %d ]\n", i % window_size, i / window_size);
		//	rtPrintf("Current absolute 2D index: [ %d, %d ]\n", idx.x, idx.y);
		//}
	}

	/*mean *= 1.f/ squared_window_size;*/
	mean = 1.f / squared_window_size * mean;

	/* compute variance */
	for (uint i = 0; i < squared_window_size; i++)
	{
		uint2 idx = make_uint2((i % window_size + top_left_window_corner.x) % screen.x, (i / window_size + top_left_window_corner.y) % screen.y);
		float3 input_buffer_val = make_float3(input_buffer[idx].x, input_buffer[idx].y, input_buffer[idx].z);
		float var = 1.f / 3.f * (input_buffer_val.x + input_buffer_val.y + input_buffer_val.z);
		/*variance += var * var;*/
		variance += (var * var - 2.0f * mean * var + mean * mean);
	}

	//variance = 1.f / squared_window_size * (variance) - (mean * mean);
	variance = 1.f / squared_window_size * variance;

	//rtPrintf("Current variance: %f\n", variance);

	return variance;
};

static __device__ __inline__ void window_test(uint2 center, uint window_size)
{
	size_t2 screen = input_buffer.size();

	float mean = 0.f;
	float variance = 0.f;
	uint squared_window_size = window_size * window_size;
	uint half_window_size = (window_size / 2) + (window_size % 2);
	uint2 top_left_window_corner = make_uint2(center.x - half_window_size, center.y - half_window_size);

	//rtPrintf("\nTop left window corner: [ %d, %d ]\n", top_left_window_corner.x, top_left_window_corner.y);

	/* compute mean value */
	for (uint i = 0; i < squared_window_size; i++)
	{
		uint2 idx = make_uint2((i % window_size + top_left_window_corner.x) % screen.x, (i / window_size + top_left_window_corner.y) % screen.y);
		if (i % window_size >= i / window_size)
		{
			//rtPrintf("Current 1D index: %d\n", i);
			//rtPrintf("Current relative 2D index: [ %d, %d ]\n", i % window_size, i / window_size);
			post_process_output_buffer[idx] = make_float4(100.0f,0.0f,100.0f,1.0f);
			//rtPrintf("Current absolute 2D index: [ %d, %d ]\n", idx.x, idx.y);
		}
	}
};

static __device__ __inline__ uint compute_samples_number(float variance)
{
	uint samples_number = static_cast<uint>(clamp(static_cast<float>(variance * max_per_launch_idx_ray_budget), 0.0f, static_cast<float>(max_per_launch_idx_ray_budget)));
	return samples_number;
};

static __device__ __inline__ void write_additional_samples_number(uint2 window_center, uint window_size, uint samples_number)
{
	uint half_window_size = (window_size / 2) + (window_size % 2 * 1);
	uint squared_window_size = window_size * window_size;
	uint2 upper_top_left_window = make_uint2(window_center.x - half_window_size, window_center.y - half_window_size);
	for (size_t i = 0; i < squared_window_size; i++)
	{
		uint2 idx = make_uint2(static_cast<uint>(i / window_size) + upper_top_left_window.x, static_cast<uint>(i % window_size) + upper_top_left_window.y);
		//additional_rays_buffer[idx] = make_uchar4(samples_number, samples_number, samples_number, samples_number);
	}
};

static __device__ __inline__ void compute_sample_num_map(uint window_size)
{
	uint additional_samples_number = 0;

	size_t2 screen = input_buffer.size();

	uint modulo_width = screen.x % window_size;
	uint modulo_height = screen.y % window_size;

	uint horizontal_padding = static_cast<uint>((screen.x - modulo_width) / 2);
	uint vertical_padding = static_cast<uint>((screen.x - modulo_width) / 2);

	uint2 window_center = make_uint2(0, 0);

	uint half_window_size = (window_size / 2) + (window_size % 2 * 1);

	for (size_t i = 0; i < modulo_width * modulo_height; i++)
	{
		window_center.x = horizontal_padding + half_window_size + i / modulo_width * window_size;
		window_center.y = vertical_padding + half_window_size + i % modulo_height * window_size;
		
		float variance = compute_window_variance(window_center, window_size);

		/* actually compute 'additional_samples_number' */

		/* write 'additional_samples_number' into according window of 2D buffer 'additional_rays_buffer' */
		write_additional_samples_number(window_center, window_size, 0);
	}

};

static __device__ __inline__ uint compute_current_samples_number(uint2 current_launch_index, uint window_size) 
{
	uint sample_number = 0;

	uint additional_samples_number = 0;

	size_t2 screen = input_buffer.size();

	uint times_width = screen.x / window_size;
	uint times_height = screen.y / window_size;

	uint horizontal_padding = static_cast<uint>((screen.x - (times_width * window_size)) / 2);
	uint vertical_padding = static_cast<uint>((screen.y - (times_height * window_size)) / 2);

	uint half_window_size = (window_size / 2) + (window_size % 2);

	uint2 times_launch_index = make_uint2(((current_launch_index.x / window_size) * window_size) % screen.x, ((current_launch_index.y / window_size) * window_size) % screen.y);

	uint2 current_window_center = make_uint2(times_launch_index.x + horizontal_padding + half_window_size, times_launch_index.y + vertical_padding + half_window_size);

	float variance = compute_window_variance(current_window_center, window_size);

	sample_number = compute_samples_number(10.0f * variance);

	//rtPrintf("\nCurrent launch index: [ %d, %d ]\n", current_launch_index.x, current_launch_index.y);
	//rtPrintf("Modulo launch index: [ %d, %d ]\n", modulo_launch_index.x, modulo_launch_index.y);
	//rtPrintf("Current window center: [ %d, %d ]\n", current_window_center.x, current_window_center.y);
	//rtPrintf("Current variance: %f\n", variance);
	if (sample_number >= max_per_launch_idx_ray_budget)
	{
		rtPrintf("Current samples number: %d\n\n", sample_number);
	}

	return sample_number;
};

static __device__ __inline__ void compute_current_window_test(uint2 current_launch_index, uint window_size)
{
	uint sample_number = 0;

	uint additional_samples_number = 0;

	size_t2 screen = input_buffer.size();

	uint times_width = screen.x / window_size;
	uint times_height = screen.y / window_size;

	uint horizontal_padding = static_cast<uint>((screen.x - (times_width * window_size)) / 2);
	uint vertical_padding = static_cast<uint>((screen.y - (times_height * window_size)) / 2);

	uint half_window_size = (window_size / 2) + (window_size % 2);

	uint2 times_launch_index = make_uint2(((current_launch_index.x / window_size) * window_size) % screen.x, ((current_launch_index.y / window_size) * window_size) % screen.y);

	uint2 current_window_center = make_uint2(times_launch_index.x + horizontal_padding + half_window_size, times_launch_index.y + vertical_padding + half_window_size);

	window_test(current_window_center, window_size);
};

RT_PROGRAM void pathtrace_camera_adaptive()
{
	size_t2 screen = input_buffer.size();

	float2 inv_screen = 1.0f / make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

	float2 jitter_scale = inv_screen / sqrt_num_samples;
	unsigned int adaptive_samples_per_pixel = compute_current_samples_number(launch_index, 19);//max_per_launch_idx_ray_budget;//static_cast<unsigned int>(additional_rays_buffer[launch_index].x);
	unsigned int current_samples_per_pixel = adaptive_samples_per_pixel;
	float3 result = make_float3(0.0f);

	unsigned int adaptive_sqrt_num_samples = sqrtf(static_cast<float>(adaptive_samples_per_pixel));

	if (!adaptive_sqrt_num_samples)
	{
		++adaptive_sqrt_num_samples;
	}

	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);

	float3 pixel_color = make_float3(input_buffer[launch_index]);

	if (current_samples_per_pixel)
	{
		do
		{
			//
			// Sample pixel using jittering
			//
			unsigned int x = adaptive_samples_per_pixel % adaptive_sqrt_num_samples;
			unsigned int y = adaptive_samples_per_pixel / adaptive_sqrt_num_samples;
			float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
			float2 d = pixel + jitter*jitter_scale;
			float3 ray_origin = eye;
			float3 ray_direction = normalize(d.x*U + d.y*V + W);

			// Initialze per-ray data
			PerRayData_pathtrace prd;
			prd.result = make_float3(0.f);
			prd.attenuation = make_float3(1.f);
			prd.countEmitted = true;
			prd.done = false;
			prd.seed = seed;
			prd.depth = 0;

			// Each iteration is a segment of the ray path.  The closest hit will
			// return new segments to be traced here.
			for (;;)
			{
				Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(top_object, ray, prd);

				if (prd.done)
				{
					// We have hit the background or a luminaire
					prd.result += prd.radiance * prd.attenuation;
					break;
				}

				// Russian roulette termination 
				if (prd.depth >= rr_begin_depth)
				{
					float pcont = fmaxf(prd.attenuation);
					if (rnd(prd.seed) >= pcont)
						break;
					prd.attenuation /= pcont;
				}

				prd.depth++;
				prd.result += prd.radiance * prd.attenuation;

				// Update ray data for the next path segment
				ray_origin = prd.origin;
				ray_direction = prd.direction;
			}

			result += prd.result;
			seed = prd.seed;
		} while (--current_samples_per_pixel);

		pixel_color = result / (adaptive_sqrt_num_samples*adaptive_sqrt_num_samples);

		if (adaptive_samples_per_pixel > 1)
		{
			pixel_color = make_float3(100.0f, 0.0f, 100.0f);
		}
	}
	//
	// Update the output buffer
	//

	float a = 1.0f / (float)frame_number;
	float3 old_color = make_float3(input_buffer[launch_index]);
	post_process_output_buffer[launch_index] = make_float4(lerp(old_color, pixel_color, a), 1.0f);

	//compute_current_window_test(launch_index, 5);
}

//
// Adaptive version of pathtracing end
//

RT_PROGRAM void diffuse()
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;

    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    optix::Onb onb( ffnormal );
    onb.inverse_transform( p );
    current_prd.direction = p;

    // NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
    // with cosine density.
    current_prd.attenuation = current_prd.attenuation * diffuse_color;
    current_prd.countEmitted = false;

    //
    // Next event estimation (compute direct lighting).
    //
    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        ParallelogramLight light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - hitpoint);
        const float3 L     = normalize(light_pos - hitpoint);
        const float  nDl   = dot( ffnormal, L );
        const float  LnDl  = dot( light.normal, L );

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f )
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                const float A = length(cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                result += light.emission * weight;
            }
        }
    }

    current_prd.radiance = result;
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void miss()
{
    current_prd.radiance = bg_color;
    current_prd.done = true;
}


